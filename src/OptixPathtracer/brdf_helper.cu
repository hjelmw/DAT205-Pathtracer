#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include <optix.h>
#include <optix_math.h>

#include "common.h"
#include "random.h"
#include "helpers.h"





//////////////////////////////// //////////////////////////////// 
//
//			Helper functions for brdf, pdf and wi evaluation
//
//						**********************
//								TODO
//
//						Fix linear blending!!
//
//						**********************
// 
//////////////////////////////// //////////////////////////////// 


//////////////////////////////// //////////////////////////////// 
//
//						Material parameters
//
//////////////////////////////// //////////////////////////////// 


rtDeclareVariable(float3, Kd, , );          // Diffuse
rtDeclareVariable(float3, Ks, , );			// Specular
rtDeclareVariable(float3, Kr, , );			// Reflective
rtDeclareVariable(float3, Ka, , );			// Ambient
rtDeclareVariable(float, phong_exp, , );	// Phong lol
rtDeclareVariable(float, Pm, , );			// Metalness
rtDeclareVariable(float, Pr, , );			// Shininess
rtDeclareVariable(float, Ps, , );			// Fresnel
rtDeclareVariable(float3, Tf, , );			// Transparency
rtDeclareVariable(float3, Ke, , );			// Emissive
rtDeclareVariable(int, Kd_mapped, , );	    // Has textures



//////////////////////////////// //////////////////////////////// 
//
//						BRDF Evaluation
// 
//////////////////////////////// //////////////////////////////// 


// Simple diffuse brdf
RT_CALLABLE_PROGRAM float3 diffuse_f(float3 wi, float3 wo, float3 n)
{

	if (dot(wi, n) <= 0.0f)
		return make_float3(0.0f);
	else if (!(signbit(dot(wi, n)) == signbit(dot(wo, n))))
		return make_float3(0.0f);
	else
		return (1.0f / M_PIf) * Kd;

}


// Simple blinn phong brdf
RT_CALLABLE_PROGRAM float3 blinnphong_f(float3 wi, float3 wo, float3 n)
{
	//return make_float3(1.0f);

	float3 blinnphong_reflection_brdf;
	float3 blinnphong_refraction_brdf;
	float3 blinnphong_brdf;

	if (dot(n, wi) <= 0.0f)
		return make_float3(0.0f);

	float3 wh = normalize(wi + wo);

	float WhWi = abs(dot(wh, wi));
	float NWh = dot(n, wh);
	float NWi = dot(n, wi);
	float WoWh = dot(wo, wh);
	float NWo = dot(n, wo);

	float F = Ps + ((1.0f - Ps) * powf(1.0f - WhWi, 5.0f));

	if (Pr <= 0.0f)
		return make_float3(0.0f);

	float s = powf(NWh, Pr);
	float a2 = Pr + 2.0f;
	float D = (a2 / (2.0f * M_PIf)) * s;

	if (WoWh <= 0.0f)
		return make_float3(0.0f);

	float m1 = 2 * (NWh * NWo / WoWh);
	float m2 = 2 * (NWh * NWi / WoWh);
	float G = min(1.0f, min(m1, m2));

	if (NWo <= 0 || NWi <= 0)
		return make_float3(0.0f);
		//blinnphong_reflection_brdf = make_float3(0.0f);

	// Blinn Phong Reflection BRDF
	blinnphong_reflection_brdf = make_float3(F * D * G / (4 * NWo * NWi));

	//return blinnphong_reflection_brdf;

	float3 diffuse_brdf = diffuse_f(wo, wi, n);

	blinnphong_refraction_brdf = make_float3(1.0f - Ps) * diffuse_brdf;

	// Complete Blinn Phong BRDF
	blinnphong_brdf = blinnphong_reflection_brdf + blinnphong_refraction_brdf;
	return blinnphong_brdf;
}

// blinn phong with added calculations for metalness material parameter
RT_CALLABLE_PROGRAM float3 blinnphongmetal_f(float3 wi, float3 wo, float3 n)
{
	float3 blinnphongmetal_brdf = make_float3(0.0f);

	float3 blinnphong_reflection_brdf;
	float3 blinnphong_refraction_brdf;
	float3 blinnphong_brdf;

	if (dot(n, wi) <= 0.0f)
		return make_float3(0.0f);

	float3 wh = normalize(wi + wo);


	float WhWi = abs(dot(wh, wi));
	float NWh = dot(n, wh);
	float NWi = dot(n, wi);
	float WoWh = dot(wo, wh);
	float NWo = dot(n, wo);

	float F = Ps + ((1.0f - Ps) * powf(1.0f - WhWi, 5.0f));

	float s = powf(NWh, Pr);
	float a2 = Pr + 2.0f;
	float D = (a2 / (2.0f * M_PIf)) * s;

	if (WoWh <= 0.0f)
		return make_float3(0.0f);

	float m1 = 2 * (NWh * NWo / WoWh);
	float m2 = 2 * (NWh * NWi / WoWh);
	float G = min(1.0f, min(m1, m2));

	if (NWo <= 0 || NWi <= 0)
		return make_float3(0.0f);

	blinnphongmetal_brdf = make_float3(F * D * G / (4 * NWo * NWi));

	return blinnphongmetal_brdf * Kd;
}

// linear blend brdf between reflective and metalness blinn phong
RT_CALLABLE_PROGRAM float3 linearblend_f(float3 wi, float3 wo, float3 n)
{

	float3 metal_blend = Pm * blinnphongmetal_f(wi, wo, n) + (1.0f -Pm) * blinnphong_f(wi, wo, n);
	float3 reflectivity_blend = Ks * metal_blend + (1.0f - Ks) * diffuse_f(wi, wo, n);

	//return metal_blend;
	return reflectivity_blend;

}



//////////////////////////////// //////////////////////////////// 
//
//						Importance sampling
// 
//////////////////////////////// //////////////////////////////// 


// Samples a new wi direction for pathtracing and returns a simple diffuse brdf
RT_CALLABLE_PROGRAM float3 diffuse_samplewi(unsigned int seed, float3& wi, const float3& wo, const float3& n, float& p)
{
	float3 tangent;
	float3 bitangent;

	create_onb(n, tangent, bitangent);

	float z1 = rnd(seed);
	float z2 = rnd(seed);
	float3 sample;
	optix::cosine_sample_hemisphere(z1, z2, sample);

	wi = normalize(sample.x * tangent + sample.y * bitangent + sample.z * n);
	if (dot(wi, n) <= 0.0f)
		p = 0.0f;
	else
		p = max(0.0f, dot(n, wi)) / M_PIf;
	return diffuse_f(wi, wo, n);
}

// Samples a new wi direction for pathtracing and returns a simple blinnphong brdf
RT_CALLABLE_PROGRAM float3 blinnphong_samplewi(unsigned int seed, float3& wi, const float3& wo, const float3& n, float& p)
{
	float pdf;
	float bdf_val;
	float3 tangent;
	float3 bitangent;

	// Importance sample
	create_onb(n, tangent, bitangent);
	float2 sample = make_float2(rnd(seed), rnd(seed));

	float3 wh = sample_phong_lobe(sample, Pr, tangent, bitangent, n);

	if (dot(wo, n) <= 0.0f)
		return make_float3(0.0f);

	if (rnd(seed) < 0.5f)
	{
		wi = normalize(-wo + 2 * dot(wh, wo) * wh); // reflect wo around wh
		//wi = reflect(-wo, wh);
		float p_wh = (Pr + 1) * pow(dot(n, wh), Pr) / (2 * M_PIf);
		float p_wi = p_wh / (4 * dot(wo, wh));

		p = p_wi;
		p = p * 0.5f;

		// Reflection BRDF
		float3 blinnphong_reflection_brdf;

		if (dot(n, wi) <= 0.0f)
			return make_float3(0.0f);

		float3 wh = normalize(wi + wo);

		float WhWi = abs(dot(wh, wi));
		float NWh = dot(n, wh);
		float NWi = dot(n, wi);
		float WoWh = dot(wo, wh);
		float NWo = dot(n, wo);

		float F = Ps + ((1.0f - Ps) * powf(1.0f - WhWi, 5.0f));

		float s = powf(NWh, Pr);
		float a2 = Pr + 2.0f;
		float D = (a2 / (2.0f * M_PIf)) * s;

		if (WoWh == 0.0f)
			return make_float3(0.0f);

		float m1 = 2 * (NWh * NWo / WoWh);
		float m2 = 2 * (NWh * NWi / WoWh);
		float G = min(1.0f, min(m1, m2));

		if (NWo == 0 || NWi == 0)
			return make_float3(0.0f);
		//blinnphong_reflection_brdf = make_float3(0.0f);

		// Blinn Phong Reflection BRDF
		blinnphong_reflection_brdf = make_float3(F * D * G / (4 * NWo * NWi));
		
		return blinnphong_reflection_brdf;
	}
	else
	{
		float3 brdf = diffuse_samplewi(seed, wi, wo, n, p);
		p = p * 0.5f;
		float F = Ps + (1.0f - Ps) * pow(1.0f - abs(dot(wh, wi)), 5.0f);
		return (1 - F) * brdf;
	}
}

// Samples a new wi direction for pathtracing and returns a simple blinnphong brdf
RT_CALLABLE_PROGRAM float3 blinnphongmetal_samplewi(unsigned int seed, float3& wi, const float3& wo, const float3& n, float& p)
{
	float pdf;
	float bdf_val;
	float3 tangent;
	float3 bitangent;

	// Importance sample
	create_onb(n, tangent, bitangent);
	float2 sample = make_float2(rnd(seed), rnd(seed));

	float3 wh = sample_phong_lobe(sample, Pr, tangent, bitangent, n);

	if (dot(wo, n) <= 0.0f)
		return make_float3(0.0f);

	if (rnd(seed) < 0.5f)
	{
		wi = normalize(-wo + 2 * dot(wh, wo) * wh); // reflect wo around wh
		float p_wh = (Pr + 1) * pow(dot(n, wh), Pr) / (2 * M_PIf);
		float p_wi = p_wh / (4 * dot(wo, wh));

		p = p_wi;
		p = p * 0.5f;

		// Reflection BRDF
		float3 blinnphong_reflection_brdf;

		if (dot(n, wi) <= 0.0f)
			return make_float3(0.0f);

		float3 wh = normalize(wi + wo);

		float WhWi = abs(dot(wh, wi));
		float NWh = dot(n, wh);
		float NWi = dot(n, wi);
		float WoWh = dot(wo, wh);
		float NWo = dot(n, wo);

		float F = Ps + ((1.0f - Ps) * powf(1.0f - WhWi, 5.0f));

		if (Pr <= 0.0f)
			return make_float3(0.0f);

		float s = powf(NWh, Pr);
		float a2 = Pr + 2.0f;
		float D = (a2 / (2.0f * M_PIf)) * s;

		if (WoWh <= 0.0f)
			return make_float3(0.0f);

		float m1 = 2 * (NWh * NWo / WoWh);
		float m2 = 2 * (NWh * NWi / WoWh);
		float G = min(1.0f, min(m1, m2));

		if (NWo <= 0 || NWi <= 0)
			return make_float3(0.0f);
		//blinnphong_reflection_brdf = make_float3(0.0f);

		// Blinn Phong Reflection BRDF
		blinnphong_reflection_brdf = make_float3(F * D * G / (4 * NWo * NWi));

		return blinnphong_reflection_brdf * Kd;
	}
	else
	{
		p = p * 0.5f;
		return make_float3(0.0f);
	}
}



// Samples a new wi direction for pathtracing and returns a brdf linearly blended between reflectivity and metalness
RT_CALLABLE_PROGRAM float3 linearblend_samplewi(unsigned int seed, float3& wi, const float3& wo, const float3& n, float& p)
{
	p = 0.0f;

	// Reflectivity
	// Ks - average reflectivity
	if (rnd(seed) < (Ks.x + Ks.y + Ks.z)/3)
	{
		// Metalness
		if (rnd(seed) < Pm)
		{
			return blinnphongmetal_samplewi(seed, wi, wo, n, p);
		}
		else
		{
			return blinnphong_samplewi(seed, wi, wo, n, p);
		}
	}
	else
	{
		return diffuse_samplewi(seed, wi, wo, n, p);
	}
}