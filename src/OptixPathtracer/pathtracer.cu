#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

#include <optix.h>
#include <optix_math.h>

#include "helpers.h"
#include "common.h"



 //////////////////////////////// //////////////////////////////// 
 //
 //			Ray gen, closest hit, any hit (shadow) programs
 //				Also see brdf_helper.cu for helper functions
 // 
 //////////////////////////////// //////////////////////////////// 

#include "brdf_helper.cu"

//////////////////////////////// //////////////////////////////// 



using namespace optix;


// Variables from 
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


// Ray tracing variables and structs for storing results
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );
rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(float, scene_epsilon, , );


// BVH declaration
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );


// Le scene lights
rtBuffer<BasicLight> lights;
rtTextureSampler<float4, 2> envmap;


// Pinhole camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(Matrix3x3, normal_matrix, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


// Buffers for storing intermediary and final output pixel value
rtBuffer<float4, 2>              output_buffer;
rtBuffer<float4, 2>              accum_buffer;
rtBuffer<float4, 2>              input_albedo_buffer;
rtBuffer<float4, 2>              input_normal_buffer;

// Ray gen variables
rtDeclareVariable(unsigned int, max_depth, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, frame_number, , );



RT_PROGRAM void trace_paths()
{
	//
	// Per thread variables
	//
	const size_t2 screen = output_buffer.size();
	unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number + 1);

	const float jitter_x = rnd(seed);
	const float jitter_y = rnd(seed);
	const float2 jitter = make_float2(jitter_x, jitter_y);
	const float2 d = (make_float2(launch_index) + jitter) / make_float2(launch_dim) * 2.f - 1.f;

	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x * U + d.y * V + W);


	//
	// Initialze per-ray data
	//
	PerRayData_pathtrace prd;
	prd.result = make_float3(0.f);
	prd.attenuation = make_float3(1.0f);
	prd.done = false;
	prd.next_done = false;
	prd.seed = seed;
	prd.depth = 0;


	//
	// Create initial ray that we fire from the camera
	//
	Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
	rtTrace(top_object, ray, prd);


	//
	// Store resulting color
	//
	prd.result = prd.radiance;


	//
	// Update our result
	//
	float3 result = prd.result;
	float3 albedo = prd.albedo;
	float3 normal_eyespace = (length(prd.normal) > 0.f) ? normalize(normal_matrix * prd.normal) : make_float3(0., 0., 1.);
	float3 normal = normal_eyespace;
	seed = prd.seed;


	//
	// Update the output buffer
	//
	float3 pixel_color = result;
	float3 pixel_albedo = albedo;
	float3 pixel_normal = normal;
	if (frame_number > 1)
	{
		float a = 1.0f / (float)frame_number;
		float3 old_color = make_float3(output_buffer[launch_index]);
		float3 old_albedo = make_float3(input_albedo_buffer[launch_index]);
		float3 old_normal = make_float3(input_normal_buffer[launch_index]);
		output_buffer[launch_index] = make_float4(lerp(old_color, pixel_color, a));
		input_albedo_buffer[launch_index] = make_float4(lerp(old_albedo, pixel_albedo, a), 1.0f);


		// this is not strictly a correct accumulation of normals, but it will do for this sample
		float3 accum_normal = lerp(old_normal, pixel_normal, a);
		input_normal_buffer[launch_index] = make_float4((length(accum_normal) > 0.f) ? normalize(accum_normal) : pixel_normal, 1.0f);

	}
	else
	{
		output_buffer[launch_index] = make_float4(pixel_color);
		input_albedo_buffer[launch_index] = make_float4(pixel_albedo, 1.0f);
		input_normal_buffer[launch_index] = make_float4(pixel_normal, 1.0f);
	}

}



//-----------------------------------------------------------------------------
//
//  Closest hit program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_li()
{

	//
	// Initialize variables
	//
	float3 result = make_float3(0.0f);
	float3 path_throughput = make_float3(1.0f);

	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hit_point = ray.origin + t_hit * ray.direction;

	
	// The albedo buffer should contain an approximation of the overall surface albedo (i.e. a single
	// color value approximating the ratio of irradiance reflected to the irradiance received over the
	// hemisphere). This can be approximated for very simple materials by using the diffuse color of
	// the first hit.

	if (current_prd.depth == 0)
	{
		current_prd.albedo = Kd;
		current_prd.normal = ffnormal;
	}


	//
	// Next event estimation (compute direct lighting).
	//
	unsigned int num_lights = lights.size();


	//
	// Light constants (currently we only have one light)
	//
	BasicLight light = lights[0];
	float3 light_pos = light.position;
	const float  Ldist = length(light_pos - hit_point);
	const float3 L = normalize(light_pos - hit_point);


	//
	// Store resulting shadow attenuation in this struct
	// 
	PerRayData_pathtrace_shadow shadow_prd;
	shadow_prd.inShadow = false;


	//
	// Shoot a shadow ray
	//
	Ray shadow_ray = make_Ray(hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon);
	rtTrace(top_shadower, shadow_ray, shadow_prd);


	//
	// Direct lighting contribution
	//
	if (!shadow_prd.inShadow)
	{
		const float distance_to_light = length(light_pos - hit_point);
		const float falloff_factor = 1.0f / (distance_to_light * distance_to_light);
		float3 Li = 2500.0f * light.color * falloff_factor;
		float3 wi = normalize(light_pos - hit_point);

		// Add direct light contribution
		//
		// Evaluate f using
		// 1. diffuse_f
		// 2. blinnphong_f
		// 3. linearblend_f

		result += linearblend_f(wi, -ray.direction, ffnormal) * Li * max(0.0f, dot(wi, ffnormal));
	}


	//
	// Add emissive lighting (e.g ship cannons)
	//
	result += path_throughput * Ke * Kd;

	float pdf;
	float3 wi;


	// Evaluate brdf, pdf and sample new wi for redirecting next ray
	//
	// Select from
	// 1. diffuse_samplewi
	// 2. blinnphong_samplewi
	// 3. linearblend_samplewi

	float3 brdf = linearblend_samplewi(current_prd.seed, wi, -ray.direction, ffnormal, pdf);
	float cosine_term = abs(dot(wi, ffnormal));


	//
	// If pdf becomes too small we reuturn to avoid NaN values
	//
	if (pdf < scene_epsilon)
	{
		current_prd.radiance = result;
		current_prd.done = true;
		return;
	}


	//
	// Math
	//
	path_throughput = path_throughput * (brdf * cosine_term) / pdf;
	

	//
	// If attenuation becomes too small we need to break
	//
	if (path_throughput.x == 0 &&
		path_throughput.y == 0 &&
		path_throughput.z == 0 )
	{
		current_prd.radiance = result;
		current_prd.done = true;
		return;
	}


	//
	// Redirect our next ray according to what we sampled, this gives pretty reflections
	//
	current_prd.origin = hit_point;
	current_prd.direction = wi;


	//
	// Avoid pesky recursion max depth issues
	//
	if (current_prd.depth < max_depth)
	{

		//
		// Create reflection ray
		//
		Ray reflection_ray = make_Ray(current_prd.origin, current_prd.direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);

		//
		// Store reflection result in separate struct
		//
		PerRayData_pathtrace reflection_prd;


		//
		// Before recursing we need to pass it the currently accumulated color (and traversal depth)
		//
		reflection_prd.attenuation = path_throughput;
		reflection_prd.radiance = result;
		reflection_prd.depth = current_prd.depth + 1;


		//
		// Shoot the reflection ray (recursion time)
		//
		rtTrace(top_object, reflection_ray, reflection_prd);


		//
		// Add resulting color
		//
		result  += reflection_prd.radiance;

	}

	//
	// We are done
	//
	current_prd.radiance = result;
}



//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

// Set pixel to solid color upon failure
RT_PROGRAM void exception()
{
	// Something went wrong? Pink pixel
	output_buffer[launch_index] = make_float4(255, 0, 255, 255);
}



//-----------------------------------------------------------------------------
//
//  Environment Map Miss program
//
//-----------------------------------------------------------------------------
// !! Environment map is defined in brdf_helper.cu !!

RT_PROGRAM void envmap_miss()
{
	const float theta = acos(max(-1.0f, min(1.0f, -ray.direction.y)));
	float phi = atan2f(ray.direction.z, ray.direction.x);
	if (phi < 0.0f)
		phi = phi + 2.0f * M_PIf;
	float2 lookup = make_float2(phi / (2.0 * M_PIf), theta / M_PIf);

	current_prd.radiance = make_float3(tex2D(envmap, lookup.x, lookup.y)) * current_prd.attenuation;

	//current_prd.radiance = make_float3(0.0f, 0.0f, 0.0);
	//reflection_prd.radiance = make_float3(0.4f, 0.0f, 0.0f);

	current_prd.done = true;
	//reflection_prd.done = true;

	if (current_prd.depth == 0)
	{
		current_prd.albedo = make_float3(0, 0, 0);
		current_prd.normal = make_float3(0, 0, 0);
	}
}


RT_PROGRAM void miss()
{
	current_prd.done = true;
	current_prd.radiance = make_float3(0.0f);
	current_prd.attenuation = make_float3(0.0f);
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

RT_PROGRAM void shadow()
{
	current_prd_shadow.inShadow = true;
	//current_prd.attenuation = make_float3(0.0f);
	//current_prd.radiance = make_float3(0.0f);
	rtTerminateRay();
}
